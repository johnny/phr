#include "hip/hip_runtime.h"
/* Necessary changes to parallelize the vanilla version with cuda

   (1) copy all subroutines for generating, I/O and time measurement to here
       (mainly necessary to avoid complicating the Makefile).
   (2) make it compile with C++ compiler (as nvcc uses that)
   (3) replace double3 by float4, replace all x[][3] by float4*
   (4) store mass in fourth component of position
   (5) use nvcc
   (6) float4 is struct in CUDA
   (7) remove symmetry optimization

 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <omp.h>

// typedef float float4[4]; // provided by NVIDIA

/*const double gamma = 6.674E-11;*/
const float G = 1.0;
const float epsilon2 = 1E-5;

// number of threads in one block
#define BLOCKSIZE 128

// kernel to be executed on the GPU
__global__ void acceleration_kernel (int jstart, float4 *x, float4 *a)
{
	extern __shared__ float4 xcol[];
    int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x+tid;

    // load column positions an mass in shared memory, each thread loads one element 
	xcol[tid] = x[jstart+tid];
	__syncthreads();

	// compute contribution to body i
	float4 xi = x[i];  // load own position
	float4 ai = a[i];  // place to accumulate result for own body
  	float3 d;
	float r,r2,factorj,invfact;
	for (int j=0; j<blockDim.x; j++)
      {
		d.x = xcol[j].x-xi.x;
		d.y = xcol[j].y-xi.y;
		d.z = xcol[j].z-xi.z;
		r2 = d.x*d.x + d.y*d.y + d.z*d.z + epsilon2;
		r = sqrtf(r2);
		invfact = G/(r*r2);
		factorj = xcol[j].w*invfact;
		ai.x += factorj*d.x;
		ai.y += factorj*d.y;
		ai.z += factorj*d.z;
      }

    // write back result
    a[i] = ai;
}


void acceleration (int n, float4 *x, float4 *a)
{
    int size = n*sizeof(float4);

    // allocate x in global memory on the device
    float4 *xd;
    hipMalloc( (void**) &xd, size ); // allocate memory on device
    hipMemcpy(xd,x,size,hipMemcpyHostToDevice); // copy x to device
    if( hipGetLastError() != hipSuccess) 
    {
        fprintf(stderr,"error in memcpy\n");
        exit(-1);
    }                         

    // allocate a in global memory on the device
	// note: a has already been cleared
    float4 *ad;
    hipMalloc( (void**) &ad, size ); // allocate memory on device
    hipMemcpy(ad,a,size,hipMemcpyHostToDevice); // copy a to device
    if( hipGetLastError() != hipSuccess) 
    {
        fprintf(stderr,"error in memcpy\n");
        exit(-1);
    }                         

	// determine block and grid size
	dim3 dimBlock(BLOCKSIZE);   // use BLOCKSIZE threads in one block
    dim3 dimGrid(n/BLOCKSIZE);  // use enough blocks to reach n; n is a multiple of BLOC

	// process chunks of rows
	for (int j=0; j<n; j+=BLOCKSIZE)
    {
        acceleration_kernel<<<dimGrid,dimBlock,BLOCKSIZE*sizeof(float4)>>>(j,xd,ad);
  		hipDeviceSynchronize();
    }

	// read result
	hipMemcpy(a,ad,size,hipMemcpyDeviceToHost);
    if( hipGetLastError() != hipSuccess) 
    {
        fprintf(stderr,"error in memcpy\n");
        exit(-1);
    }                         

	// free memory on device
	hipFree(xd);
	hipFree(ad);
}

void acceleration_cpu (int n, float4 *x, float4 *a)
{
  int i,j;
  float d0,d1,d2,r,r2,factorj,invfact;

  /* compute acceleration exploiting symmetry */
  for (i=0; i<n; i++)
    for (j=0; j<n; j++)
      {
		d0 = x[j].x-x[i].x;
		d1 = x[j].y-x[i].y;
		d2 = x[j].z-x[i].z;
		r2 = d0*d0 + d1*d1 + d2*d2 + epsilon2;
		r = sqrt(r2);
		invfact = G/(r*r2);
		factorj = x[j].w*invfact;
		a[i].x += factorj*d0;
		a[i].y += factorj*d1;
		a[i].z += factorj*d2;
      }
}

void leapfrog (int n, float dt, float4 *x, float4 *v, float4 *a, float4 *aold)
{
  int i;
  float dt2 = dt*dt*0.5;
  float dthalf = dt*0.5;

  /* update position */
  for (i=0; i<n; i++)
    {
      x[i].x += dt*v[i].x + dt2*a[i].x;
      x[i].y += dt*v[i].y + dt2*a[i].y;
      x[i].z += dt*v[i].z + dt2*a[i].z;
    }

  /* save and clear acceleration */
  for (i=0; i<n; i++)
    {
      aold[i].x = a[i].x;
      aold[i].y = a[i].y;
      aold[i].z = a[i].z;
      a[i].x = a[i].y = a[i].z = 0.0;
    }
  
  /* compute new acceleration */
  acceleration(n,x,a);

  /* update velocity */
  for (i=0; i<n; i++)
    {
      v[i].x += dthalf*aold[i].x + dthalf*a[i].x;
      v[i].y += dthalf*aold[i].y + dthalf*a[i].y;
      v[i].z += dthalf*aold[i].z + dthalf*a[i].z;
    }
}


void write_vtk_file_float (FILE *f, int n, float4 *x, float4 *v,
			    float t, float dt)
{
  int i;

  /* header */
  fprintf(f,"%s\n","# vtk DataFile Version 1.0");
  fprintf(f,"NBODY %22.16g %22.16g\n",t,dt);
  fprintf(f,"%s\n","ASCII");

  /* points */
  fprintf(f,"%s\n","DATASET POLYDATA");
  fprintf(f,"%s %d %s\n","POINTS",n,"float");
  for (i=0; i<n; i++)
    fprintf(f,"%22.16g %22.16g %22.16g\n",x[i].x,x[i].y,x[i].z);

  /* vertices */
  fprintf(f,"%s %d %d\n","VERTICES",n,2*n);
  for (i=0; i<n; i++)
    fprintf(f,"%d %d\n",1,i);

  /* scalar data fields*/
  fprintf(f,"%s %d\n","POINT_DATA",n);
  fprintf(f,"%s\n","SCALARS mass float");
  fprintf(f,"%s\n","LOOKUP_TABLE default");
  for (i=0; i<n; i++)
    fprintf(f,"%22.16g\n",x[i].w);

  /* vector data */
  fprintf(f,"%s\n","VECTORS velocity float");
  for (i=0; i<n; i++)
    fprintf(f,"%22.16g %22.16g %22.16g\n",v[i].x,v[i].y,v[i].z);
}
    
void cube (int n, long int seed, float size, float m0, float mdelta,  
		float4 *x, float4 *v)
{
  int i;
  float3 s;
  float3 t;
  float M = 0.0;

  s.x = s.y = s.z = 0.0;
  t.x = t.y = t.z = 0.0;

  if (seed!=0) srand48(seed);
  for (i=0; i<n; i++)
    {
      x[i].x = drand48()*size;
      x[i].y = drand48()*size;
      x[i].z = drand48()*size;
      v[i].x = 0.0;
      v[i].y = 0.0;
      v[i].z = 0.0;
      x[i].w = m0 + (drand48()-0.5)*2.0*mdelta;
      s.x += x[i].w*x[i].x;
      s.y += x[i].w*x[i].y;
      s.z += x[i].w*x[i].z;
      t.x += x[i].w*v[i].x;
      t.y += x[i].w*v[i].y;
      t.z += x[i].w*v[i].z;
      M += x[i].w;
    }
  printf("center of mass: %g %g %g\n",s.x,s.y,s.z);
  for (i=0; i<n; i++)
    {
      x[i].x -= s.x; 
      x[i].y -= s.y; 
      x[i].z -= s.z; 

      v[i].x -= t.x; 
      v[i].y -= t.y; 
      v[i].z -= t.z; 
    }
}

void plummer (int n, long int seed,
			  float4 *x, float4 *v)
{
  /* This is a copy from 
     Pit Hut, Jun Makino: The Art of Computational Science, 
     The Kali Code, vol. 5. Initial Conditions: Plummer's Model.
  */ 
  int i;
  float radius,theta,phi,X,Y,velocity,maxr=-1.0;
  const float Pi = 3.141592653589793238462643383279;
  float3 s;
  float3 t;
  s.x = s.y = s.z = 0.0;
  t.x = t.y = t.z = 0.0;

  if (seed!=0) srand48(seed);
  for (i=0; i<n; i++)
    {
      x[i].w = 1.0/n;
      radius = 1.0/sqrt(pow(drand48(),-2.0/3.0)-1.0);
      if (radius>maxr) maxr=radius;
      theta = acos(-1.0+drand48()*2.0);
      phi = drand48()*2.0*Pi;
      x[i].x = radius*sin(theta)*cos(phi);
      x[i].y = radius*sin(theta)*sin(phi);
      x[i].z = radius*cos(theta);
      s.x += x[i].w*x[i].x;
      s.y += x[i].w*x[i].y;
      s.z += x[i].w*x[i].z;
      X = 0.0;
      Y = 0.1;
      while (Y>X*X*pow(1.0-X*X,3.5))
	{
	  X = drand48();
	  Y = drand48()*0.1;
	}
      velocity = X*sqrt(2.0)*pow(1.0+radius*radius,-0.25);
      theta = acos(-1.0+drand48()*2.0);
      phi = drand48()*2.0*Pi;
      v[i].x = velocity*sin(theta)*cos(phi);
      v[i].y = velocity*sin(theta)*sin(phi);
      v[i].z = velocity*cos(theta);
      t.x += x[i].w*v[i].x;
      t.y += x[i].w*v[i].y;
      t.z += x[i].w*v[i].z;
    }
  printf("center of mass: %g %g %g\n",s.x,s.y,s.z);
  for (i=0; i<n; i++)
    {
      x[i].x -= s.x; 
      x[i].y -= s.y; 
      x[i].z -= s.z; 

      v[i].x -= t.x; 
      v[i].y -= t.y; 
      v[i].z -= t.z; 
    }
  s.x=s.y=s.z=0.0;
  for (i=0; i<n; i++)
    {
      s.x += x[i].w*x[i].x;
      s.y += x[i].w*x[i].y;
      s.z += x[i].w*x[i].z;
    }
  printf("new center of mass: %g %g %g\n",s.x,s.y,s.z);
  printf("maximum radius: %g\n",maxr);
}

float get_time ()
{
/*   struct rusage ru; */
/*   struct timeval cstop; */
  
/*   getrusage(RUSAGE_SELF, &ru); */
/*   cstop = ru.ru_utime; */
/*   return 1.0*cstop.tv_sec + cstop.tv_usec/1000000.0; */
/*   return clock()/CLOCKS_PER_SEC; */
  return omp_get_wtime();
}

int main (int argc, char** argv)
{
  int n;
  float4 *x;
  float4 *v;
  float4 *a;
  float4 *temp;
  int i,k,mod;
  char base[]="test1";
  char name[256];
  FILE *file;
  float t,dt;
  float start,stop;
  float elapsed,flop;
  int timesteps;

  if (argc!=4)
	{
	  printf("usage: nbody_cuda <nbodies> <timesteps> <every>\n");
	  return 1;
	}
  sscanf(argv[1],"%d",&n);
  sscanf(argv[2],"%d",&timesteps);
  sscanf(argv[3],"%d",&mod);

    // determine block size and grid
    if (n%BLOCKSIZE != 0)
    {
        printf("n must be a multiple of BLOCKSIZE\n");
		exit(-1);
    }

  /* allocate memory, read data file */
  /*   file = fopen("plummer10000.vtk","r"); */
  /*   if (file==NULL) */
  /*     { */
  /*       printf("could not open file --- aborting\n"); */
  /*       return 1; */
  /*     } */
  /*   n = get_vtk_numbodies(file); */
  /*   rewind(file); */
  /*   x = calloc(n,sizeof(float4)); */
  /*   v = calloc(n,sizeof(float4)); */
  /*   m = calloc(n,sizeof(float)); */
  /*   read_vtk_file_float(file,n,x,v,m,&t,&dt); */
  /*   fclose(file); */
  /*   printf("loaded %d bodies\n",n); */
  x = (float4*) calloc(n,sizeof(float4));
  v = (float4*)calloc(n,sizeof(float4));
  plummer(n,17,x,v);

  a = (float4*) calloc(n,sizeof(float4));
  temp = (float4*) calloc(n,sizeof(float4));
  for (i=0; i<n; i++)
    a[i].x = a[i].y = a[i].z = 0.0;
  acceleration(n,x,a);

  k = 0;
  t = 0.0;
  dt = 1E-3;
  printf("writing %s_%06d.vtk \n",base,k);
  sprintf(name,"%s_%06d.vtk",base,k);
  file = fopen(name,"w");
  write_vtk_file_float(file,n,x,v,t,dt);
  fclose(file);
  start = get_time();

  for (k=1; k<timesteps; k++)
    {
      leapfrog(n,dt,x,v,a,temp);
      t += dt;
      if (k%mod==0)
		{
		  stop = get_time();
		  elapsed = stop-start;
		  flop = mod*(19.0*n*n+24.0*n);
		  printf("%g seconds for %g ops = %g MFLOPS\n",elapsed,flop,flop/elapsed/1E6);
		  printf("writing %s_%06d.vtk \n",base,k/mod);
		  
		  sprintf(name,"%s_%06d.vtk",base,k/mod);
		  file = fopen(name,"w");
		  write_vtk_file_float(file,n,x,v,t,dt);
		  fclose(file);
		  
		  start = get_time();
		}
    }

  return 0;
}
